#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void square(int* array, int n) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    printf("%d ", tid);
    if (tid < n) { // checks for thread index out of bounds
        array[tid] = array[tid] * array[tid];
    }
}

int main() {
    const int n = 1000;
    int* a = new int[n];
    for (int i = 0; i < n; i++) {
        a[i] = i;
    }

    int* dev_a;
    size_t size = n * sizeof(int);

    // Allocate memory on the device
    hipError_t err = hipMalloc(&dev_a, size);
    if (err != hipSuccess) {
        printf("Error in cudaMalloc: %s\n", hipGetErrorString(err));
        return 1;
    }

    // Copy data from host to device
    err = hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("Error in hipMemcpy (HostToDevice): %s\n", hipGetErrorString(err));
        hipFree(dev_a);
        return 1;
    }

    // Determine the number of blocks and threads
    int threadsPerBlock = 256;  // arbitrarily picking 256 threads per block
    //int blocks = n / threadsPerBlock; // THIS IS WRONG: rounds down
    int blocks = (n + threadsPerBlock - 1) / threadsPerBlock; // round up to the min number of blocks needed

    printf("thread ids:");
    // Launch kernel with the correct number of blocks and threads
    square<<<blocks, threadsPerBlock>>>(dev_a, n);
    printf("\n======\n");

// is the kernel call syncronized automatically?
#if 0
    // Synchronize to make sure kernel execution is finished
    err = cudaDeviceSynchronize();
    if (err != cudaSuccess) {
        printf("Error in kernel execution: %s\n", cudaGetErrorString(err));
        cudaFree(dev_a);
        return 1;
    }
#endif
    // Copy the results back to host
    err = hipMemcpy(a, dev_a, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("Error in hipMemcpy (DeviceToHost): %s\n", hipGetErrorString(err));
        hipFree(dev_a);
        return 1;
    }

    printf("The squares are:\n");
    // Print the first 64 elements
    for (int i = 0; i < n; i++) {
        printf("%d ", a[i]);
    }
    printf("\n");

    // Clean up
    hipFree(dev_a);
    delete[] a;

    return 0;
}
